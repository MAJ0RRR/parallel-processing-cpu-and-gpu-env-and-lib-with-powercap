#include "hip/hip_runtime.h"
/*
Copyright 2023 Paweł Czarnul pczarnul@eti.pg.edu.pl

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files (the “Software”), to deal in the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
sell copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/
// idea: in most cases, input data can be uploaded to GPU's memory and
// consequently we only need to copy a pointer in kernel invocation
// in OpenCL we could hide any kernel invocation

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define ENABLE_LOGGING_GPU
#define ENABLE_LOGGING
#include "logger_gpu.h"
#include "logger.h"
#include "vecmaxdiv_defines.h"

__global__ void appkernel(void *devPtr) {
  double *devPtra = (double *)(((void **)devPtr)[0]);
  double *devPtrb = (double *)(((void **)devPtr)[1]);
  double *devPtrc = (double *)(((void **)devPtr)[2]);

  long my_index = blockIdx.x * blockDim.x + threadIdx.x;
  long i;
  long result = 1;

  long max = sqrt(devPtra[my_index]);
  long elem = devPtra[my_index];
  for (i = 2; i < max; i++) {
    if (!(elem % i)) {
      if (i >= result) {
        result = i;
      }
    }
  }

  max = sqrt(devPtrb[my_index]);
  elem = devPtrb[my_index];
  for (i = 2; i < max; i++) {
    if (!(elem % i)) {
      if (i >= result) {
        result = i;
      }
    }
  }

  devPtrc[my_index] = result;
}

extern "C" void launchkernelinstream(void *devPtr, hipStream_t stream) {

  dim3 blocksingrid(VECMAXDIV_BLOCKS_IN_GRID);
  dim3 threadsinblock(VECMAXDIV_THREADS_IN_BLOCK);

  log_message(LOG_DEBUG, "Launichng GPU Kernel with %i blocks in grid and %i threads in block.", VECMAXDIV_BLOCKS_IN_GRID, VECMAXDIV_THREADS_IN_BLOCK);
  appkernel<<<blocksingrid, threadsinblock, 0, stream>>>(devPtr);

  if (hipSuccess != hipGetLastError()) {
    log_message(LOG_ERROR, "Error during kernel launch in stream");
  }
}

extern "C" void launchkernel(void *devPtr) { launchkernelinstream(devPtr, 0); }
